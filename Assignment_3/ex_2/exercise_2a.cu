#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <cstdlib>
#include <ctime>
#include <chrono>
#include <cmath>

struct Particle {
	float3 position;
	float3 velocity;
};

bool compareParticles(Particle *particles1, Particle *particles2, int size);

void updateParticlesCPU(Particle *particles, int size, float velocityGiven);

void initializeParticles(Particle *particles, int arraySize);

__global__ void UPDATE_PARTICLES(Particle *particles, const float velocityGiven, const int arraySize)
{

	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < arraySize) {
		particles[i].velocity.x = velocityGiven * (i + 1);
		particles[i].position.x = particles[i].position.x + particles[i].velocity.x * 1;
	}

}

int main()
{
	const int NUM_PARTICLES = 40000;
	const int NUM_ITERATIONS = 1000;
	const int BLOCK_SIZE = 256;

	const int BLOCKS = (NUM_PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE;

	Particle particles[NUM_PARTICLES] = {};


	const float randomVelocity = rand();
	float d2 = rand();

	printf("%d\n", fabs(0.2) < fabs(0.5) ? 1 : 0);


	initializeParticles(particles, NUM_PARTICLES);

	Particle *particles_GPU = 0;
	hipError_t cudaStatus;

	printf("Starting GPU particle simulation now\n");

	//Time before GPU runs update
	auto current_time = std::chrono::system_clock::now();
	auto duration_in_seconds = std::chrono::duration<double>(current_time.time_since_epoch());
	double gpu_before = duration_in_seconds.count();


	//cudaStatus = hipMalloc((void**)&particles_GPU, NUM_PARTICLES * sizeof(Particle));
	//cudaStatus = hipHostMalloc((void**)&particles_GPU, NUM_PARTICLES * sizeof(Particle));
	cudaStatus = hipMallocManaged(&particles_GPU, NUM_PARTICLES * sizeof(Particle));

	cudaStatus = hipMemcpy(particles_GPU, particles, NUM_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("malloc failed\n");
		return 1;
	}


	//TIMESTEP
	for (int i = 0; i < NUM_ITERATIONS; i++) {

		//Update
		UPDATE_PARTICLES << <BLOCKS, BLOCK_SIZE >> > (particles_GPU, randomVelocity, NUM_PARTICLES);

		hipDeviceSynchronize();

		//work with data here
		Particle temp  = particles_GPU[1];
		particles[1] = particles[0];
		particles[0] = temp;
		temp = particles[1];
		particles[1] = particles[0];
		particles[0] = temp;

	}

	hipDeviceSynchronize();

	if (cudaStatus != hipSuccess)
	{
		hipHostFree(particles_GPU);
		return 1;
	}

	current_time = std::chrono::system_clock::now();
	duration_in_seconds = std::chrono::duration<double>(current_time.time_since_epoch());
	double gpu_after = duration_in_seconds.count();

	printf("GPU Finished in time:%f\n", (gpu_after - gpu_before));


	current_time = std::chrono::system_clock::now();
	duration_in_seconds = std::chrono::duration<double>(current_time.time_since_epoch());
	double CPU_before = duration_in_seconds.count();

	for (int i = 0; i < NUM_ITERATIONS; i++) {
		updateParticlesCPU(particles, NUM_PARTICLES, randomVelocity);
	}

	current_time = std::chrono::system_clock::now();
	duration_in_seconds = std::chrono::duration<double>(current_time.time_since_epoch());
	double CPU_after = duration_in_seconds.count();

	printf("CPU Finished in time:%f\n", (CPU_after - CPU_before));


	printf("Comparing particles...");

	bool result = compareParticles(particles_GPU, particles, NUM_PARTICLES);
	if (result == true) {
		printf(" Correct!\n");
	}
	else {
		printf(" Not the same...\n");
	}

	hipFree(particles_GPU);


	return 0;

}




bool compareParticles(Particle *particles1, Particle *particles2, int size) {

	for (int i = 0; i < size; i++) {
		if (fabs(particles1[i].position.x - particles2[i].position.x) > 0.01 || fabs(particles1[i].velocity.x - particles2[i].velocity.x) > 0.01)
			return false;
	}
	return true;
}


void updateParticlesCPU(Particle *particles, int size, float velocityGiven) {
	for (int i = 0; i < size; i++) {
		particles[i].velocity.x = velocityGiven * (i + 1);
		particles[i].position.x = particles[i].position.x + particles[i].velocity.x * 1;
	}


}



void initializeParticles(Particle *particles, int arraySize) {

	for (int i = 0; i < arraySize; i++) {
		particles[i].velocity.x = rand();
		particles[i].velocity.y = rand();
		particles[i].velocity.z = rand();
		particles[i].position.x = rand();
		particles[i].position.y = rand();
		particles[i].position.z = rand();
	}

}
